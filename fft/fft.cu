#include "fft.h"
#include "fftexceptions.h"
#include "UtilNPP/Exceptions.h"
#include "UtilNPP/ImageIO.h"
#include "UtilNPP/ImagesCPU.h"
#include "UtilNPP/ImagesNPP.h"
#include <npp.h>
#include "hip/hip_runtime_api.h"
#include "helper_string.h"
#include <hip/hip_runtime.h>
#include <filesystem>
#include <iostream>

namespace fft {

void ProcessFFT(const FFTConfig& config)
{
    try {
        std::cout << __PRETTY_FUNCTION__ << std::endl;
    } catch (const npp::Exception& e) {
        throw fft::FFTException(e.toString());
    }
}

}